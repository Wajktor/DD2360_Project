#include <hip/hip_runtime.h>

#include <iostream>

class Managed {
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

struct particle2 : public Managed{
	int id;
	int len;
	char *name;
};

struct particle{
	int id;
	int len;
	float *nums;
};


void allocate(struct particle **par){

	// **par = &oldpar
	hipMallocManaged( &(*par), sizeof(particle));

	hipMallocManaged( &((*par)->nums), sizeof(float) * 3 );
}

__global__ void change(struct particle *par){

	par->nums[0] = 1337.0;

}

__global__ void blesd(){

	float weight[2000][2000][2000];
	weight[0][0][0] = 99;

	for(int i = 0; i < 1; ++i){
		printf("weight: %f \n", weight[0][0][0]);
	}

}


int main(){


	blesd<<<1,1>>>();

	hipDeviceSynchronize();

	return 0;
}